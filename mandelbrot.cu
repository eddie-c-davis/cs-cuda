#include "hip/hip_runtime.h"

/************************************\
| filename: escape.c
|
| description: sequential version
| of code that outputs a .PGM file of
| a Mandelbrot fractal.
|
| notes: the number of pixels, 2400x2400
| was chosen so that it would take a fair
| amount of time to compute the image so
| that speedup may be observed on in a parallel
| implementation.  it might be advisable
| to change the #defines for the purposes
| of developing a parallel version of the
| code.
|
| hint: the file output is a .PGM file which
| is viewable with the linux utility gimp.
| The 'convert' utility can convert
| from .pgm to .gif, which will save lots of disk
| space.
|
| authors: Bryan Schlief, Daegon Kim, Wim Bohm
|
\***********************************/


#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <limits.h>

// CUDA includes...
#include <hip/hip_runtime.h>
//#include <>

#define   RMIN       0.3129928802767
#define   RMAX       0.31299305009252
#define   IMIN       0.0345483210604
#define   IMAX       0.0345485012278

#define   RADIUS_SQ  4.0     /* 2^2                              */
#define   WIDTH      2400    /* # of pixels wide                 */
#define   HEIGHT     2400    /* # of pixels high                 */
#define   MAX_COLOR  UCHAR_MAX
#define   OUT_FILE   "Mandelbrot.pgm"
#define   BLOCK_SIZE 32
#define   DEF_ITER   1000
#define   DEBUG      0

void writeOutput(const char *fileName, char *data, int width, int height) {
    int i, j;      /* index variables */
    int max = -1;  /* for pgm file output */
    int size = width * height;

    /* PGM file format requires the largest pixel value, calculate this */
    for (i = 0; i < size; ++i) {
        if (data[i] > max) {
            max = data[i];
        }
    }

    /* open the file for writing. omit error checking. */
    FILE * fout = fopen(fileName, "w");

    /* PGM file header */
    fprintf(fout, "P2\n");
    fprintf(fout, "%d\t%d\n", width, height);
    fprintf(fout, "%d\n",max);

    /* throw out the data */
    for (i = 0; i < height; ++i) {
        for (j = 0; j < width; ++j) {
            fprintf(fout, "%d\t", data[i * width + j]);
        }

        fprintf(fout,"\n");
    }

    /* flush the buffer and close the file */
    fflush(fout);
    fclose(fout);
}

#define cudaAssert(ans) { _cudaAssert((ans), __FILE__, __LINE__); }
inline void _cudaAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess)  {
        fprintf(stderr, "cudaAssert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

void cudaPrintDevices(FILE *file, hipDeviceProp_t *prop, int i) {
    fprintf(file, "Device Number: %d\n", i);
    fprintf(file, "  Device name: %s\n", prop->name);
    fprintf(file, "  Memory Clock Rate (KHz): %d\n", prop->memoryClockRate);
    fprintf(file, "  Memory Bus Width (bits): %d\n", prop->memoryBusWidth);
    fprintf(file, "  Peak Memory Bandwidth (GB/s): %f\n", 2.0* prop->memoryClockRate * (prop->memoryBusWidth / 8) / 1.0e6);
    fprintf(file, "  Compute Version: %d.%d\n", prop->major, prop->minor);
    fprintf(file, "  Compute Mode: ");

    switch (prop->computeMode) {
        case hipComputeModeExclusive:
            fprintf(file, "Exclusive");
            break;
        case hipComputeModeProhibited:
            fprintf(file, "Prohibited");
            break;
        default:
            fprintf(file, "Default");
            break;
    }

    fprintf(file, "\n");
    fprintf(file, "  SM count: %d\n", prop->multiProcessorCount);
    fprintf(file, "  Shared mem/block: %zd\n", prop->sharedMemPerBlock);
    fprintf(file, "  Threads per warp: %d\n", prop->warpSize);
    fprintf(file, "  Max threads per block: %d\n", prop->maxThreadsPerBlock);

    fprintf(file, "  Max block size: (");
    for (int j = 0; j < 3; j++) {
        fprintf(file, "%d,", prop->maxThreadsDim[j]);
    }

    fprintf(file, ")\n  Max grid size: (");
    for (int j = 0; j < 3; j++) {
        fprintf(file, "%d,", prop->maxGridSize[j]);
    }

    fprintf(file, ")\n\n");

}

__global__ void mand(char* output, int maxIter, double realRange, double imagRange) {
    int col = blockDim.x * blockIdx.x + threadIdx.x;  // Image col (X coord)
    int row = blockDim.y * blockIdx.y + threadIdx.y;  // Image row (Y coord)

    if (col < WIDTH && row < HEIGHT) {
        int idx = row * WIDTH + col;

        double cReal = RMIN + row * realRange;
        double cImag = IMIN + col * imagRange;

        double zReal = 0.0;
        double zImag = 0.0;
        double zReal2 = zReal;
        double zImag2 = zImag;
        double zCurr;
        double zMag;

        int iter = 0;
        for (; iter < maxIter; ++iter) {
            zCurr = zReal;

            zReal2 = zReal * zReal;
            zImag2 = zImag * zImag;

            zReal = zReal2 - zImag2 + cReal;
            zImag = (2.0 * zCurr * zImag) + cImag;

            zMag = zReal2 + zImag2;
            if (zMag > RADIUS_SQ) {
                break;
            }
        }

        output[idx] = (char) floor(((double) (MAX_COLOR * iter)) / (double) maxIter);
    }
}

int main(int argc, char ** argv) {
    int nDevices = 0;
    hipDeviceProp_t prop;

    char *hostOutput = NULL;
    char *devOutput = NULL;

    float time; 	/*timer*/

    int maxIter = DEF_ITER;
    if (argc > 1) {
        maxIter = atoi(argv[1]);    /* first command line argument... */
    }

    if (maxIter < 1) {
        printf("Usage : %s [MAX ITERATION]\n", argv[0]);
        return 0;
    }

    printf("Running Mandelbrot-CUDA with %d iterations...\n", maxIter);

    cudaAssert(hipGetDeviceCount(&nDevices));

    if (nDevices < 1) {
        printf("ERROR: No valid CUDA devices on this machine!\n");
        return -1;
    }

    if (DEBUG) {
        fprintf(stderr, "nDevices = %d\n", nDevices);
        for (int i = 0; i < nDevices; i++) {
            cudaAssert(hipGetDeviceProperties(&prop, i));
            cudaPrintDevices(stderr, &prop, i);
        }
    }

    // Get data size...
    int dataSize = WIDTH * HEIGHT;
    if (DEBUG) fprintf(stderr, "dataSize = %d\n", dataSize);

    /* Allocate memory on host to store output values for pixels */
    hostOutput = (char *) calloc(dataSize, sizeof(char));
    if (hostOutput == NULL) {
        perror("hostOutput");
        return -1;
    }

    // Set block size...
    int blockWidth = BLOCK_SIZE;
    int blockHeight = blockWidth;
    dim3 blockSize(blockWidth, blockHeight);
    if (DEBUG) fprintf(stderr, "blockSize = (%d,%d)\n", blockSize.x, blockSize.y);

    // Set grid size...
    int gridWidth = WIDTH / blockSize.x;
    int gridHeight = HEIGHT / blockSize.y;
    dim3 gridSize(gridWidth, gridHeight);
    if (DEBUG) fprintf(stderr, "gridSize = (%d,%d)\n", gridSize.x, gridSize.y);

    // Create event timers...
    hipEvent_t start, stop;
    cudaAssert(hipEventCreate(&start));
    cudaAssert(hipEventCreate(&stop));

    // Start timer...
    hipEventRecord(start);

    /* Allocate memory on device... */
    if (DEBUG) fprintf(stderr, "hipMalloc...\n");
    cudaAssert(hipMalloc(&devOutput, dataSize * sizeof(char)));

    double realRange = (RMAX - RMIN) / (double) (WIDTH - 1);
    double imagRange = (IMAX - IMIN) / (double) (HEIGHT - 1);

    // Invoke the kernel...
    if (DEBUG) {
        fprintf(stderr, "kernel: mand(devOutput[%d], maxIter=%d, realRange=%lf, imagRange=%lf)...\n",
                dataSize, maxIter, realRange, imagRange);
    }

    mand<<<gridSize, blockSize>>>(devOutput, maxIter, realRange, imagRange);

    // Check last error...
    if (DEBUG) fprintf(stderr, "hipPeekAtLastError...\n");
    cudaAssert(hipPeekAtLastError());

    // Sync the device...
    //if (DEBUG) fprintf(stderr, "hipDeviceSynchronize...\n");
    //cudaAssert(hipDeviceSynchronize());

    // Copy data back to host
    if (DEBUG) fprintf(stderr, "hipMemcpy...\n");
    cudaAssert(hipMemcpy(hostOutput, devOutput, dataSize, hipMemcpyDeviceToHost));

    // Free data on device...
    if (DEBUG) fprintf(stderr, "hipFree...\n");
    cudaAssert(hipFree(devOutput));

    // Stop timer...
    cudaAssert(hipEventRecord(stop));

    // Get elapsed time...
    if (DEBUG) fprintf(stderr, "hipEventSynchronize...\n");
    cudaAssert(hipEventSynchronize(stop));
    if (DEBUG) fprintf(stderr, "hipEventElapsedTime...\n");
    cudaAssert(hipEventElapsedTime(&time, start, stop));

    // Write the output...
    if (DEBUG) fprintf(stderr, "writeOutput...\n");
    writeOutput(OUT_FILE, hostOutput, WIDTH, HEIGHT);

    // Free host data...
    free(hostOutput);

    // Report timing...
    printf("Elapsed time: %lf ms\n", time);

    return 0;
}
